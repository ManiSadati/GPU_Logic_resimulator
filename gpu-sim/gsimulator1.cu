#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<iostream>
#include "../object.h"

using namespace std;


static map<int,pair<int,int> > signal_address;

extern vector<int8_t> t_ans;
extern vector<int8_t> t_edge;
extern map<string,int> t_mp;
extern vector<vector<int> > height;
extern vector<vector<int> > dealloc_signal;
extern vector<Gate> component_vector;
extern vector<Signal> signal_vector;
extern long long int MX;
extern long long int MN;
extern unordered_map<string, int> mpsignal;
extern unordered_map <int, bool> signal_mark;


static unsigned int *d_M_EVENT_time;
static int *d_M_EVENT_val;

static long long int max_part_number;
static int thread_number, sig_part_number;
static int gpu_size;


class Buddy{
public:
	const int max_n = 40;
	set<long long int > S[100];
	map<int,long long int> mp;
	Buddy(){
		S[max_n].insert(0);
	}
	long long int find_buddy(long long int r,long long int level){
		return r^(1LL<<level);
	}
	long long int t_allocate(int size){
		long long int k=log2(size);
		if((1LL<<k)!=size)
		k++;
		long long int pos=k;
		for(int i=k;i<100;i++){
			if(S[i].size()!=0){
				pos=i;
				break;
			}
		}

		for(int i=pos;i>k;i--){
			long long int node=*S[i].begin();
			S[i].erase(S[i].begin());
			long long int left=node;
			long long int right=find_buddy(node,i-1);
			S[i-1].insert(left);
			S[i-1].insert(right);
			mp[left]=i-1;
			mp[right]=i-1;
		}
		if(S[k].size()!=0){
			long long int ans=*S[k].begin();
			S[k].erase(S[k].begin());
			return ans;
		}
		return -1;
	}
	void t_deallocate(long long int pos){
		int p=mp[pos];
		while(true){
			long long int bud=find_buddy(pos,p);
			auto fin=S[p].find(bud);
			if(fin!=S[p].end()){
				S[p].erase(fin);
				//int x=*fin;
				p++;
				pos=min(bud,pos);
			}
			else
			break;
		}
		mp[pos]=p;
		S[p].insert(pos);
	}

};

static Buddy tree;

static int total_number_of_gates, total_number_of_ports, total_number_of_partitions, total_number_of_port_per_parts, total_sum_of_sdf, total_number_of_signal_parts, total_input_sig;

static int *h_sig_input_M_pointer, *d_sig_input_M_pointer;
static int *h_sig_input_D_pointer, *d_sig_input_D_pointer;
static int *h_sig_input_size, *d_sig_input_size;
static unsigned int *h_input_time, *d_input_time;
static int *h_input_val, *d_input_val;

static int *h_gate_start, *d_gate_start;
static int *h_gate_end, *d_gate_end;
static int *h_port_start, *d_port_start;
static int *h_port_end, *d_port_end;

static int *h_gate_ind, *d_gate_ind;
static int *h_gate_partition_number, *d_gate_partition_number;
static int *d_out_pointer;

static int *h_gate_partition_index, *d_gate_partition_index;
static int *d_port_partition_start;
static int *d_port_partition_end;
static int *d_partition_pre;
static int *d_partition_address_first, *d_partition_address_last;
static bool *d_zero_time;
static int *h_partition_number, *d_partition_number;
static int *h_stimuli_sdf_first, *d_stimuli_sdf_first;
static int *h_stimuli_sdf_end, *d_stimuli_sdf_end;
static int *h_table_id, *d_table_id;
static int *h_table, *d_table;
static unsigned int *d_tmp_memory_time;
static int *h_partition_output_address, *d_partition_output_address;
static unsigned int *d_partition_min_time;
static int *h_stimuli_size, *d_stimuli_size;
static int *h_sdf_mask, *d_sdf_mask;
static int *h_sdf_s_in, *d_sdf_s_in;
static int8_t *h_t_edge , *d_t_edge;
static int *h_signal_pointer_first, *d_signal_pointer_first;
static int *h_signal_pointer_last, *d_signal_pointer_last;
static bool *h_signal_ending_part, *d_signal_ending_part;
static unsigned int *h_saif_part, *d_saif_part;
static bool *h_fake_signal, *d_fake_signal;
static bool *h_fake_part, *d_fake_part;


void input_init_g1(){

	hipMalloc(&d_M_EVENT_time, gpu_size*sizeof(unsigned int));
	hipMalloc(&d_M_EVENT_val, gpu_size*sizeof(int));
	int total_event = 0;
	for(int i = 0 ; i < signal_vector.size(); i++){
		Signal& signall = signal_vector[i];
		int id = mpsignal[signall.name];
		if(id != i) continue;
		if(signal_mark[i] == false && signall.vaweform.size() > 0){
			signall.netlist_input = true;
		}
		if(signall.netlist_input == false)continue;
		total_input_sig ++;
		total_event += signall.vaweform.size();
	}

	h_sig_input_M_pointer = (int *)malloc(total_input_sig*sizeof(int));
	h_sig_input_D_pointer = (int *)malloc(total_input_sig*sizeof(int));
	h_sig_input_size = (int *)malloc(total_input_sig*sizeof(int));
	h_input_time = (unsigned int *)malloc(total_event*sizeof(unsigned int));
	h_input_val = (int *)malloc(total_event*sizeof(int));

	hipMalloc(&d_sig_input_M_pointer, total_input_sig*sizeof(int));
	hipMalloc(&d_sig_input_D_pointer, total_input_sig*sizeof(int));
	hipMalloc(&d_sig_input_size, total_input_sig*sizeof(int));
	hipMalloc(&d_input_time, total_event*sizeof(unsigned int));
	hipMalloc(&d_input_val, total_event*sizeof(int));

	int cnt_input_sig = 0;
	int cnt_event = 0;
	for(int i = 0 ; i < signal_vector.size(); i++){
		Signal& signall = signal_vector[i];
		int id = mpsignal[signall.name];
		if(id != i) continue;
		int lev = signall.last_level;
		dealloc_signal[lev].push_back(i);
		if(signall.netlist_input == false)continue;
		int sz = signall.vaweform.size();
		int ptr1 = tree.t_allocate(sz);
		int ptr2 = ptr1 + sz;
		if(ptr1 > gpu_size - gpu_size / 8 || ptr2 > gpu_size){
			cout<<"sdfsdfsdfsdfsdf "<<signall.name<<" "<<ptr1<<" "<<ptr2<<endl;
		}
		signall.alloc_ptr_first = ptr1;
		if(ptr1 < 0 || ptr1 >= gpu_size){
			cout<<"sshohoh"<<endl;
			exit(0);
		}
		signall.alloc_ptr_last = ptr2;
		//cout<<"allocate "<<signall.name<<" "<<ptr1<<" "<<ptr2<<" "<<sz<<endl;
		h_sig_input_M_pointer[cnt_input_sig] = ptr1;
		h_sig_input_D_pointer[cnt_input_sig] = cnt_event;
		h_sig_input_size[cnt_input_sig] = ptr2 - ptr1;
		for(int j = 0 ; j < signall.vaweform.size(); j++){
			long long int next_time = MX;
			long long int this_time = MN;
			int val = signall.vaweform[j].value;
			if(j + 1 < signall.vaweform.size()){
				next_time = max(MN, min(MX, signall.vaweform[j+1].time));
			}
			this_time = max(MN, min(MX, signall.vaweform[j].time));
			if(val == 0 ){
				signall.o_val += next_time - this_time;
			}
			if(val == 1 ){
				signall.i_val += next_time - this_time;
			}
			if(val >= 2 ){
				signall.x_val += next_time - this_time;
			}
			h_input_time[cnt_event] = signall.vaweform[j].time;
			h_input_val[cnt_event] = signall.vaweform[j].value;
			cnt_event++;
		}

		//cout<<"saif : "<<signall.name<<" "<<signall.o_val<<" "<<signall.i_val<<" "<<signall.x_val<<endl;
		cnt_input_sig++;
	}

	hipMemcpy(d_sig_input_M_pointer, h_sig_input_M_pointer, total_input_sig*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sig_input_D_pointer, h_sig_input_D_pointer,total_input_sig*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sig_input_size, h_sig_input_size,total_input_sig*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_input_time, h_input_time,total_event*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_input_val, h_input_val,total_event*sizeof(int), hipMemcpyHostToDevice);

}

void input_free_g1(){
	free(h_sig_input_M_pointer);
	free(h_sig_input_D_pointer);
	free(h_sig_input_size);
	free(h_input_time);
	free(h_input_val);

	hipFree(d_sig_input_M_pointer);
	hipFree(d_sig_input_D_pointer);
	hipFree(d_sig_input_size);
	hipFree(d_input_time);
	hipFree(d_input_val);
}

__global__ void kernel0_input_init_g1_g1(int sig_part_number,int sig_n, int *d_sig_input_M_pointer, int *d_sig_input_D_pointer, int *d_sig_input_size, unsigned int *d_input_time, int *d_input_val, unsigned int *d_M_EVENT_time, int *d_M_EVENT_val){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < sig_n * sig_part_number){
		int gate_ind = index / sig_part_number;
		int part_ind = index % sig_part_number;

		int size = d_sig_input_size[gate_ind];
		int p1 = (size * part_ind) / sig_part_number;
		int p2 = (size * (part_ind + 1)) / sig_part_number;
		if(part_ind == sig_part_number - 1){
			 p2 = size;
		}
		int t_pointer1 = d_sig_input_M_pointer[gate_ind];
		int d_pointer1 = d_sig_input_D_pointer[gate_ind];
		for(int i = p1 ; i < p2; i++){
			d_M_EVENT_val[t_pointer1 + i] = d_input_val[d_pointer1 + i];
			d_M_EVENT_time[t_pointer1 + i] = d_input_time[d_pointer1 + i];
		}
	}
}

static vector<int> gate_partition_vec;
static vector<long long int> gate_size_vec;
static long long int total_size_of_inputs;

void initialize_level_g1(int LEV){
	gate_partition_vec.clear();
	gate_size_vec.clear();

	total_size_of_inputs = 0;
	total_number_of_gates = 0;
	total_number_of_ports = 0;
	total_number_of_partitions = 0;
	total_number_of_port_per_parts = 0;
	total_number_of_signal_parts = 0;
	total_sum_of_sdf = 0;

	//deallocation
	for(int j = 0 ; j < dealloc_signal[LEV].size(); j++){
		int ind = dealloc_signal[LEV][j];
		Signal& signall = signal_vector[ind];
		if(signall.alloc_ptr_first != -1)
			tree.t_deallocate(signall.alloc_ptr_first);
		signall.alloc_ptr_first = -1;
		signall.alloc_ptr_last = -1;
	}

	cout<<height[LEV].size()<<endl;
	for(int i = 0 ; i < height[LEV].size(); i++){
		int comp_ind = height[LEV][i];
		Gate& compo = component_vector[comp_ind];
		int insz = compo.inputs.size();
		int outsz = compo.outputs.size();
		long long int input_size = 0;

		for(int j = 0 ; j < compo.inputs.size() ; j++){
			int signal_id = compo.inputs[j].signal_index;
			Signal& signall = signal_vector[signal_id];
			int ptr1 = signall.alloc_ptr_first;
			int ptr2 = signall.alloc_ptr_last;
			input_size += ptr2 - ptr1;
		}
		for(int j = 0 ; j < outsz; j++){
			gate_size_vec.push_back(input_size);
			total_size_of_inputs += input_size;
		}
	}

	for(int i = 0; i < height[LEV].size(); i++){

		int comp_ind = height[LEV][i];
		Gate& compo = component_vector[comp_ind];
		int insz = compo.inputs.size();
		int outsz = compo.outputs.size();
		long long int input_size = gate_size_vec[total_number_of_gates];

		//most important line:
		//long long int part_number = max( 1LL , (max_part_number * input_size) / total_size_of_inputs) ;
		long long int part_number = min(input_size / 100 , (max_part_number * input_size) / total_size_of_inputs) + 1;

		for(int j = 0 ; j < outsz; j++){
			gate_partition_vec.push_back(part_number);
		}

		total_sum_of_sdf += 16 * compo.dl2.size();
		total_number_of_gates += outsz;
		total_number_of_ports += insz * outsz;
		total_number_of_partitions += part_number * outsz;
		total_number_of_port_per_parts += insz * outsz * part_number;
		total_number_of_signal_parts += outsz * sig_part_number;
	}
}

void memory_allocate_g1(){

	h_gate_start = (int *)malloc(total_number_of_gates*sizeof(int));
	hipMalloc(&d_gate_start, total_number_of_gates*sizeof(int));
	h_gate_end = (int *)malloc(total_number_of_gates*sizeof(int));
	hipMalloc(&d_gate_end, total_number_of_gates*sizeof(int));

	h_port_start = (int *)malloc(total_number_of_ports*sizeof(int));
	hipMalloc(&d_port_start, total_number_of_ports*sizeof(int));
	h_port_end = (int *)malloc(total_number_of_ports*sizeof(int));
	hipMalloc(&d_port_end, total_number_of_ports*sizeof(int));

	h_gate_partition_index = (int *)malloc(total_number_of_gates*sizeof(int));
	hipMalloc(&d_gate_partition_index, total_number_of_gates*sizeof(int));

	h_gate_ind = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_gate_ind, total_number_of_partitions*sizeof(int));

	h_gate_partition_number = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_gate_partition_number, total_number_of_partitions*sizeof(int));

	hipMalloc(&d_out_pointer, total_number_of_partitions*sizeof(int));

	hipMalloc(&d_port_partition_start, total_number_of_port_per_parts*sizeof(int));
	hipMalloc(&d_port_partition_end, total_number_of_port_per_parts*sizeof(int));

	hipMalloc(&d_partition_pre, total_number_of_port_per_parts*sizeof(int));

	hipMalloc(&d_partition_address_first, total_number_of_partitions*sizeof(int));
	hipMalloc(&d_partition_address_last, total_number_of_partitions*sizeof(int));

	hipMalloc(&d_zero_time, total_number_of_partitions*sizeof(bool));

	h_partition_number = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_partition_number, total_number_of_partitions*sizeof(int));

	h_stimuli_sdf_first = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_stimuli_sdf_first, total_number_of_partitions*sizeof(int));
	h_stimuli_sdf_end = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_stimuli_sdf_end, total_number_of_partitions*sizeof(int));

	h_table_id = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_table_id, total_number_of_partitions*sizeof(int));

	h_table = (int *)malloc((t_ans.size())*sizeof(int));
	hipMalloc(&d_table, (t_ans.size())*sizeof(int));

	hipMalloc(&d_tmp_memory_time, total_number_of_partitions * 12 * sizeof(unsigned int));

	h_partition_output_address = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_partition_output_address, total_number_of_partitions*sizeof(int));

	hipMalloc(&d_partition_min_time, total_number_of_partitions*sizeof(unsigned int));

	h_stimuli_size = (int *)malloc(total_number_of_partitions*sizeof(int));
	hipMalloc(&d_stimuli_size, total_number_of_partitions*sizeof(int));

	h_sdf_mask = (int *)malloc(total_sum_of_sdf*sizeof(int));
	hipMalloc(&d_sdf_mask, total_sum_of_sdf*sizeof(int));

	h_sdf_s_in = (int *)malloc((total_sum_of_sdf / 8)*sizeof(int));
	hipMalloc(&d_sdf_s_in, (total_sum_of_sdf / 8)*sizeof(int));

	h_fake_signal = (bool *)malloc(total_number_of_partitions*sizeof(bool));
	hipMalloc(&d_fake_signal, total_number_of_partitions*sizeof(bool));

	h_t_edge = (int8_t *)malloc((t_edge.size())*sizeof(int8_t));
	hipMalloc(&d_t_edge, (t_edge.size())*sizeof(int8_t));

	h_signal_pointer_first = (int *)malloc(total_number_of_signal_parts*sizeof(int));
	hipMalloc(&d_signal_pointer_first, total_number_of_signal_parts*sizeof(int));
	h_signal_pointer_last = (int *)malloc(total_number_of_signal_parts*sizeof(int));
	hipMalloc(&d_signal_pointer_last, total_number_of_signal_parts*sizeof(int));
	h_signal_ending_part = (bool *)malloc(total_number_of_signal_parts*sizeof(bool));
	hipMalloc(&d_signal_ending_part, total_number_of_signal_parts*sizeof(bool));

	h_saif_part = (unsigned int *)malloc(total_number_of_signal_parts*3*sizeof(unsigned int));
	hipMalloc(&d_saif_part, total_number_of_signal_parts*3*sizeof(unsigned int));

	h_fake_part = (bool *)malloc(total_number_of_signal_parts*sizeof(bool));
	hipMalloc(&d_fake_part, total_number_of_signal_parts*sizeof(bool));

}


__global__ void kernel_print_waveform_g1(int ptr1, int ptr2, int *d_M_EVENT_val, unsigned int *d_M_EVENT_time){
	printf(" printing waveforme : p1 : %d  p2 : %d \n", ptr1, ptr2 );
	for(int i = ptr1; i < ptr2; i++){
		if(  ptr2  - i < 50 || i - ptr1 < 20)
		printf("- %lld %d\n",d_M_EVENT_time[i], d_M_EVENT_val[i] );
	}
}

void set_arrays_g1(int LEVEL){
	int cnt_gate = 0;
	int cnt_partition = 0;
	int cnt_sdf = 0;
	int cnt_port = 0;
	int cnt_partition_index = 0;

	for(int j = 0 ; j < t_ans.size(); j++){
		h_table[j] = (int)t_ans[j];
	}
	for(int j = 0 ; j < 16; j++){
		h_t_edge[j] = t_edge[j];
	}
	//cout<<"a"<<endl;
	for(int i = 0 ; i < height[LEVEL].size(); i++){
		int comp_ind = height[LEVEL][i];
		Gate& compo = component_vector[comp_ind];
		int output_num = compo.size - compo.l_in;
		int mask2 = pow(3, compo.l_in);
		for(int op = 0 ; op < output_num; op++){
			//cout<<"gate : "<<compo.name<<" "<<op<<" cntgate : "<<cnt_gate<<" cntport : "<<cnt_port<<" cntpart: "<<cnt_partition<<endl;
			//sdf part
			int o_sig_id = compo.outputs[op].signal_index;
			Signal& o_sig = signal_vector[o_sig_id];

			int cnt_sdf1 = cnt_sdf;
			for(int k = 0 ; k < compo.dl2.size(); k++){
					delay2& del = compo.dl2[k];
					if(del.out != compo.l_in + op) continue;

					h_sdf_s_in[cnt_sdf/ 8]= del.in;
					for(int i_edge = 1 ; i_edge < 3; i_edge++){
						for(int o_edge = 1 ; o_edge < 5; o_edge++){
							int x = ((i_edge - 1) * 4 +  (o_edge - 1)) + cnt_sdf;

							int y = 999;
							if(del.edge == 0 || del.edge == i_edge){
								if(o_edge == 1){
									y = del.ris;
								}
								else if(o_edge == 2){
									y = del.fal;
								}
								else if(o_edge == 3){
									y = max(del.ris, del.fal);
								}
								else if(o_edge == 4){
									y = min(del.ris, del.fal);
								}
							}
							h_sdf_mask[x] = y;
						}
					}
					cnt_sdf += 8;
				}
			int cnt_sdf2 = cnt_sdf;

			int part_number = gate_partition_vec[cnt_gate];
			h_gate_partition_index[cnt_gate] = cnt_partition_index;
			//cout<<"partition index : "<<cnt_port * part_number<<endl;
			h_gate_start[cnt_gate] = cnt_port;
			//cout<<cnt_gate<<" -> "<<cnt_port<<endl;
			for(int j = 0 ; j < compo.inputs.size(); j++){
				int signal_id = compo.inputs[j].signal_index;
				Signal& signall = signal_vector[signal_id];
				int ptr1 = signall.alloc_ptr_first;
				int ptr2 = signall.alloc_ptr_last;

				//if(compo.name == "u_partition_c_reset_ICCADs_sync_reset_synced_rstn_ICCADs_UI_test_mode_outmux"){
				//	cout<<"input signal "<<signall.name<<" "<<ptr1<<" "<<ptr2<<endl;
				//}
				if(ptr1 == -1 || ptr2 == -1){
					cout<<"NO WAY ptr is -1 for"<<signall.name<<endl;
					exit(0);
				}
				h_port_start[cnt_port] = ptr1;
				h_port_end[cnt_port] = ptr2;
				cnt_port++;
				cnt_partition_index += part_number;
			}
			h_gate_end[cnt_gate] = cnt_port;
			//partition part(kernel 2 and 3)
			for(int j = 0 ; j < part_number ; j++){
				h_fake_signal[cnt_partition] = false;
				if(o_sig.name == "Fake"){
					h_fake_signal[cnt_partition] = true;
				}
				h_gate_ind[cnt_partition] = cnt_gate;
				h_gate_partition_number[cnt_partition] = part_number;
				h_partition_number[cnt_partition] = j;
				h_stimuli_sdf_first[cnt_partition] = cnt_sdf1;
				h_stimuli_sdf_end[cnt_partition] = cnt_sdf2;
				h_table_id[cnt_partition] = compo.table_id + mask2 * op;
				cnt_partition++;
			}
			cnt_gate++;
		}

	}
}


__device__ int check_port_index_g1(long long int Time, int first_pointer, int last_pointer, unsigned int *d_M_EVENT_time){
	int l = first_pointer;
	int r = last_pointer ;
	while(l < r - 1){
		int mid = (r + l) / 2;
		if(d_M_EVENT_time[mid] < Time){
			l = mid;
		}
		else{
			r = mid;
		}
	}
	if(d_M_EVENT_time[l] >= Time){
		return l;
	}
	else{
		return r;
	}
}

__device__ bool check_time_g1(long long int Time, long long int tmp_size, int p1, int p2, int *portfirst, int *portlast, unsigned int *d_M_EVENT_time){
	long long int ans = 0;
	for(int i = p1 ; i < p2 ; i++){
		int index = check_port_index_g1(Time, portfirst[i - p1], portlast[i - p1], d_M_EVENT_time);
		ans += index - portfirst[i - p1] + 1;
	}
	if(ans >= tmp_size){
		return true;
	}
	return false;
}

__device__ long long int find_time_g1(long long int tmp_size, int p1, int p2, int *portfirst, int *portlast, unsigned int *d_M_EVENT_time){

	long long int l = 0;
	long long int r = 1000000LL*1000000LL;
	while(l < r - 1){
		long long int mid = (r + l) / 2;
		if(check_time_g1(mid, tmp_size, p1, p2, portfirst, portlast, d_M_EVENT_time)){
			r = mid;
		}
		else{
			l = mid;
		}
	}
	if(check_time_g1(l, tmp_size, p1, p2, portfirst, portlast, d_M_EVENT_time)){
		return l;
	}
	return r;
}

__global__ void kernel1_find_parttion_bounds_g1(int part_n, int *d_gate_partition_number, int *d_partition_number, int *d_gate_ind, bool *d_zero_time, int *d_partition_pre, int *d_partition_address_first, int *d_partition_address_last, int *d_port_partition_start, int *d_port_partition_end, int *d_gate_partition_index, int *d_gate_start, int * d_gate_end, int *d_port_start, int *d_port_end, unsigned int *d_M_EVENT_time, int *d_M_EVENT_val){

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < part_n){
		//printf("index is %d\n", index);
		long long int total_size = 0;
		int gate_ind = d_gate_ind[index];//!
		int p1 = d_gate_start[gate_ind];
		int p2 = d_gate_end[gate_ind];
		int portfirst[6];
		int portlast[6];
		for(int i = p1 ; i < p2; i++){
			portfirst[i - p1] = d_port_start[i];
			portlast[i - p1] = d_port_end[i];
		}
		for(int i = p1 ; i < p2; i++){
			total_size += portlast[i - p1] - portfirst[i - p1];
		}
		int part_number = d_gate_partition_number[index];//!
		int partitioin_pointer = d_gate_partition_index[gate_ind];
		int part_cnt = d_partition_number[index];//!
		int ptr = index - part_cnt;//!
		if(part_cnt == 0){
			d_zero_time[ptr] = true;
			//first part of first part
			for(int i = p1; i < p2; i++){
				d_port_partition_start[partitioin_pointer + i - p1] = portfirst[i - p1];
				d_partition_pre[partitioin_pointer + i - p1] = d_M_EVENT_val[portfirst[i - p1]];
			}
			d_partition_address_first[ptr] = partitioin_pointer ;
			d_partition_address_last[ptr] = partitioin_pointer + p2 - p1;

			//last of last part
			for(int j = p1 ; j < p2 ; j++){
				int ind = partitioin_pointer + ((part_number - 1) * (p2 - p1)) +  j - p1;
				d_port_partition_end[ind] = portlast[j - p1];
			}
		}
		else{
			long long int tmp_size = (total_size * part_cnt) / part_number;
			long long int new_time = find_time_g1(tmp_size, p1, p2, portfirst, portlast, d_M_EVENT_time);
			//printf("partition : %d  new time : %lld   tmpsize : %lld\n", i, new_time, tmp_size );
			d_zero_time[ptr + part_cnt] = false;
			if(new_time == 0){
				d_zero_time[ptr + part_cnt] = true;
			}
			d_partition_address_first[ptr + part_cnt] = partitioin_pointer + ((part_cnt) * (p2 - p1)) ;
			d_partition_address_last[ptr + part_cnt] = partitioin_pointer + ((part_cnt + 1) * (p2 - p1));
			for(int j = p1 ; j < p2 ; j++){
				int x = check_port_index_g1(new_time, portfirst[j - p1], portlast[j - p1], d_M_EVENT_time);
				int ind1 = partitioin_pointer + ((part_cnt - 1) * (p2 - p1)) +  j - p1;
				int ind2 = partitioin_pointer + (part_cnt * (p2 - p1)) +  j - p1;
				int z = portfirst[j - p1];
				z = max(z, x - 1);
				d_partition_pre[ind2] = d_M_EVENT_val[z];
				d_port_partition_end[ind1] = x;
				d_port_partition_start[ind2] = x;
			}
		}
		/*if(index == -1){
			for(int i = 0 ; i < part_number ; i++){
				printf("%d---partition : %d\n",index ,i );
				printf("%d  zero : %d\n",index , d_zero_time[ptr + i]);
				printf("%d  part first : %d part last : %d\n",index ,d_partition_address_first[ptr + i], d_partition_address_last[ptr + i] );
				for(int j = p1 ; j < p2 ; j++){
					int ind1 = partitioin_pointer + (i * (p2 - p1)) +  j - p1;
					printf("%d     port %d\n",index ,j );
					printf("%d       part pre : %d \n",index ,d_partition_pre[ind1] );
					printf("%d       portstart: %d portlast : %d pre part : %d\n",index , d_port_partition_start[ind1], d_port_partition_end[ind1], d_partition_pre[ind1]);
					printf("%d       index1 : %d\n",index , ind1);
					printf("%d       start Time : %lld\n",index ,d_M_EVENT_time[d_port_partition_start[ind1]] );
				}
			}

		}*/

	}
}

__global__ void kernel2_find_partition_sizes_g1(int thread_number, int part_n, int *d_out_pointer, int *d_gate_partition_number, long long int mx,int *d_partition_number, bool *d_zero_time, unsigned int *d_partition_min_time, int *d_stimuli_size, int *d_partition_pre, int *d_stimuli_sdf_first, int *d_stimuli_sdf_end, int *d_partition_address_first, int *d_partition_address_last, int *d_port_partition_start, int*d_port_partition_end, int *d_sdf_mask, int *d_sdf_s_in, int8_t *d_t_edge, int *d_table_id, unsigned int *d_tmp_memory_time, int *d_table, int *d_M_EVENT_val, unsigned int *d_M_EVENT_time){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int index0 = blockIdx.x * blockDim.x;

	__shared__ int8_t sh_table[35556];

	if(index - index0 < 32)
	{
		int chunk_sz = 35556 / 32 + 1;
		int sh_p1 = chunk_sz * (index - index0);
		int sh_p2 = min(35556, chunk_sz * (index - index0 + 1));
		for(int i = sh_p1 ; i < sh_p2 ; i++){
			sh_table[i] = d_table[i];
		}
	}
	/*if(index == index0){
		for(int i = 0; i < 35556; i++) {
			sh_table[i] = d_table[i];
		}
	}*/
	__syncthreads();

	if(index < part_n){
		int8_t sh_edge[16];
		for(int i = 0 ; i < 16; i++){
			sh_edge[i] = d_t_edge[i];
		}

		int loop_size = 12;
		int loop_ptr = index * loop_size;
		int out1 = 0;
		int stimuli_size = 0;
		unsigned int Time = 0;
		unsigned int min_partition_time = mx + 1;
		int p1 = d_partition_address_first[index];
		int p2 = d_partition_address_last[index];
		int sdf1 = d_stimuli_sdf_first[index];
		int sdf2 = d_stimuli_sdf_end[index];
		int t_id = d_table_id[index];
		int pre_in_ports[6];
		int now_in_ports[6];
		int output_val = 0;
		int n_output_val = 0;
		int last_port[6];
		int end_port[6];
		unsigned int last_time[6];
		int tmpmask = 0;
		for(int i = p1; i < p2 ; i++){
			last_port[i - p1] = d_port_partition_start[i];
			end_port[i - p1] = d_port_partition_end[i];
			pre_in_ports[i - p1] = d_partition_pre[i];
			last_time[i - p1] = d_M_EVENT_time[last_port[i - p1]];
			now_in_ports[i - p1] = d_partition_pre[i];
		}
		for(int i = p2 - 1; i >= p1 ; i--){
			tmpmask *= 3;
			tmpmask += min(2, now_in_ports[i - p1]);
		}
		tmpmask += t_id;
		output_val = sh_table[tmpmask];
		//printf(" but why ? at part : %d mask : %d   value : %d\n",index , tmpmask, output_val);
		if(d_zero_time[index] == true){
			min_partition_time = 0;
			stimuli_size++;
			d_tmp_memory_time[loop_ptr + out1] = 0;
			out1 = (out1 + 1) % loop_size;
		}

		while(true){
			int mask = 0;
			Time = mx + 1;
			for(int i = p1 ; i < p2; i++){
				if(last_port[i - p1] < end_port[i - p1]){
					//long long int time_ = d_M_EVENT_time[last_port[i - p1]];
					//Time = min (time_, Time);
					Time = min(last_time[i - p1], Time);
				}
			}
			if(Time > mx) break;
			for(int i = p2 - 1; i >= p1 ; i--){
				if(last_port[i - p1] < end_port[i - p1]){
					//long long int time_ = d_M_EVENT_time[last_port[i - p1]];
					//if(time_ == Time){
					//	now_in_ports[i - p1] = d_M_EVENT_val[last_port[i - p1]];
					//	last_port[i - p1]++;
					//}
					if(last_time[i - p1] == Time){
						now_in_ports[i - p1] = d_M_EVENT_val[last_port[i - p1]];
						last_port[i - p1]++;
						last_time[i - p1] = d_M_EVENT_time[last_port[i - p1]];
					}
				}
				mask *= 3;
				mask += min(2, now_in_ports[i - p1]);
			}
			mask += t_id;
			n_output_val = sh_table[mask];
			int output_edge = sh_edge[output_val * 4 + n_output_val];
			if(output_val != n_output_val){
				int delay_out = 999;
				for(int i = sdf1; i < sdf2; i += 8){
					int s_in = d_sdf_s_in[i / 8];
					int input_edge = sh_edge[pre_in_ports[s_in] * 4 + now_in_ports[s_in]];
					if(input_edge > 0){
						delay_out = min(delay_out, d_sdf_mask[i + ((input_edge - 1) * 4 + (output_edge - 1))]);
					}
				}
				if(delay_out == 999){
					delay_out = 0;
				}
				if(delay_out < 0){
					printf("oh noo boiii %d %d\n",delay_out, index );
				}
				unsigned int new_time = Time + delay_out;
				min_partition_time = min(min_partition_time, new_time);
				int cnt = 0;
				while(stimuli_size && d_tmp_memory_time[loop_ptr + (out1 - 1 + loop_size) % loop_size] >= new_time){
					cnt++;
					stimuli_size--;

					out1 = (out1 - 1 + loop_size) % loop_size;
				}
				if(cnt > loop_size){
					printf("slkdjfslkdjfssssssssssss %d\n", cnt);
				}
				//if(index < 5){
				//	printf("here is new event %d -> %d time : %d\n", output_val, n_output_val, new_time);
				//}

				stimuli_size++;

				d_tmp_memory_time[loop_ptr + out1] = new_time;
				out1 = (out1 + 1) % loop_size;
			}
			output_val = n_output_val;
			for(int i = p1; i < p2 ; i++){
				pre_in_ports[i - p1] = now_in_ports[i - p1];
			}
		}
		//printf(" ok now im in partition %d and min part time is : %lld\n",index, min_partition_time );
		d_partition_min_time[index] = min_partition_time;
		d_stimuli_size[index] = stimuli_size;
		d_out_pointer[index] = out1;
		//printf("part number : %d stimuli size : %d  minparttime : %lld\n",index, stimuli_size, d_partition_min_time[index] );

	}

}


__global__ void kernel2_2_find_partition_sizes_g1(int part_n,int *d_out_pointer, int *d_gate_partition_number, long long int mx,int *d_partition_number, bool *d_zero_time, unsigned int *d_partition_min_time, int *d_stimuli_size, int *d_partition_pre, int *d_stimuli_sdf_first, int *d_stimuli_sdf_end, int *d_partition_address_first, int *d_partition_address_last, int *d_port_partition_start, int*d_port_partition_end, int8_t *d_t_edge, int *d_table_id, unsigned int *d_tmp_memory_time, int *d_table, int *d_M_EVENT_val, unsigned int *d_M_EVENT_time){
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < part_n){
		int loop_size = 12;
		int loop_ptr = index * loop_size;
		int out1 = d_out_pointer[index];
		int gate_part_n = d_gate_partition_number[index];//!
		int stimuli_size = d_stimuli_size[index];
		int part_num = d_partition_number[index];//!

		unsigned int z = mx + 1;
		for(int i = part_num + 1; i < gate_part_n; i++){
			z = min(z, d_partition_min_time[index + i - part_num]);
		}
		int cnt = 0;
		while(stimuli_size && d_tmp_memory_time[loop_ptr + (out1 - 1 + loop_size) % loop_size] >= z){
			stimuli_size--;
			cnt++;
			out1 = (out1 - 1 + loop_size) % loop_size;
		}
		if(cnt > loop_size){
			printf("slk222222djfslkdjfssssssssssss %d %d %lld \n", cnt, index, z);
		}
		d_stimuli_size[index] = stimuli_size;
		if(stimuli_size < 0)
		printf("MM partintion : %d   stimuli size %d  partnumber : %d\n", index, stimuli_size, part_num);
	}

}

//kernel3
__global__ void kernel3_gpu_single_gate_simulation_g1(int thread_number, int part_n, long long int mx,int *d_partition_number, int *d_partition_output_address, bool *d_zero_time, unsigned int *d_partition_min_time, int *d_stimuli_size, int *d_partition_pre, int *d_stimuli_sdf_first, int *d_stimuli_sdf_end, int *d_partition_address_first,
	 int *d_partition_address_last, int *d_port_partition_start, int *d_port_partition_end, int *d_sdf_mask, int *d_sdf_s_in, int8_t *d_t_edge, int *d_table_id, int *d_table, int *d_M_EVENT_val, unsigned int *d_M_EVENT_time, bool *d_fake_signal){

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int index0 = blockIdx.x * blockDim.x;

	__shared__ int8_t sh_table[35556];

	if(index - index0 < 32)
	{
		int chunk_sz = 35556 / 32 + 1;
		int sh_p1 = chunk_sz * (index - index0);
		int sh_p2 = min(35556, chunk_sz * (index - index0 + 1));
		for(int i = sh_p1 ; i < sh_p2 ; i++){
			sh_table[i] = d_table[i];
		}
	}
	/*if(index == index0){
		for(int i = 0; i < 35556; i++) {
			sh_table[i] = d_table[i];
		}
	}*/
	__syncthreads();

	if(index < part_n){

		bool fake_signal = d_fake_signal[index];
		int8_t sh_edge[16];
		for(int i = 0 ; i < 16; i++){
			sh_edge[i] = d_t_edge[i];
		}

		unsigned int overflow_time = 0;
	 	int out1 = d_partition_output_address[index];
		int out2 = out1 + d_stimuli_size[index];
		int out0 = out1;

		unsigned int Time = 0;
		//int part_num = d_partition_number[index];
		int p1 = d_partition_address_first[index];
		int p2 = d_partition_address_last[index];
		int sdf1 = d_stimuli_sdf_first[index];
		int sdf2 = d_stimuli_sdf_end[index];
		int t_id = d_table_id[index];
		int pre_in_ports[6];
		int now_in_ports[6];
		int output_val = 0;
		int n_output_val = 0;
		int last_port[6];
		int end_port[6];
		unsigned int last_time[6];
		int tmpmask = 0;
		for(int i = p1; i < p2 ; i++){
			last_port[i - p1] = d_port_partition_start[i];
			end_port[i - p1] = d_port_partition_end[i];
			pre_in_ports[i - p1] = d_partition_pre[i];
			now_in_ports[i - p1] = d_partition_pre[i];
			last_time[i - p1] = d_M_EVENT_time[last_port[i - p1]];
		}
		for(int i = p2 - 1; i >= p1 ; i--){
			tmpmask *= 3;
			tmpmask += min(2, now_in_ports[i - p1]);
		}
		tmpmask += t_id;
		output_val = sh_table[tmpmask];
		if(d_zero_time[index] == true && fake_signal == false){
			if(out1 != out2){
				d_M_EVENT_val[out1] = output_val;
				d_M_EVENT_time[out1] = 0;
				out1++;
			}
		}

		while(true){
			int mask = 0;
			Time = mx + 1;
			for(int i = p1 ; i < p2; i++){
				if(last_port[i - p1] < end_port[i - p1]){
					//long long int time_ = d_M_EVENT_time[last_port[i - p1]];
					//Time = min (time_, Time);
					Time = min(last_time[i - p1], Time);
				}
			}
			if(Time > mx) break;
			for(int i = p2 - 1; i >= p1 ; i--){
				if(last_port[i - p1] < end_port[i - p1]){
					//long long int time_ = d_M_EVENT_time[last_port[i - p1]];
					//if(time_ == Time){
					//	now_in_ports[i - p1] = d_M_EVENT_val[last_port[i - p1]];
					//	last_port[i - p1]++;
					//}
					if(last_time[i - p1] == Time){
						now_in_ports[i - p1] = d_M_EVENT_val[last_port[i - p1]];
						last_port[i - p1]++;
						last_time[i - p1] = d_M_EVENT_time[last_port[i - p1]];
					}
				}
				mask *= 3;
				mask += min(2, now_in_ports[i - p1]);
			}
			mask += t_id;
			n_output_val = sh_table[mask];
			int output_edge = sh_edge[output_val * 4 + n_output_val];
			if(output_val != n_output_val){
				int delay_out = 999;
				for(int i = sdf1; i < sdf2; i += 8){
					int s_in = d_sdf_s_in[i / 8];
					int input_edge = sh_edge[pre_in_ports[s_in] * 4 + now_in_ports[s_in]];
					if(input_edge > 0){
						delay_out = min(delay_out, d_sdf_mask[i + ((input_edge - 1) * 4 + (output_edge - 1)) ]);
					}
				}
				if(delay_out == 999){
					delay_out = 0;
				}
				if(delay_out < 0){
					printf("oh noo boiii %d %d\n",delay_out, index );
				}
				unsigned int new_time = Time + delay_out;

				if(out1 > out2 && overflow_time >= new_time && fake_signal == false){
					out1--;
				}
				while(out1 > out0 && out1 <= out2 && d_M_EVENT_time[out1 - 1] >= new_time && fake_signal == false){
					out1--;
				}
				if(fake_signal == false){
					if(out1 >= out2){
						overflow_time = new_time;
						out1 = out2 + 1;
					}
					else{
						d_M_EVENT_time[out1] = (unsigned int) new_time;
						d_M_EVENT_val[out1] = n_output_val;
						out1++;
					}
				}
			}
			output_val = n_output_val;
			for(int i = p1; i < p2 ; i++){
				pre_in_ports[i - p1] = now_in_ports[i - p1];
			}
		}

	}

}

void tree_allocations_g1(int LEVEL){
	int cnt_partition = 0;
	int cnt_gate = 0;

	for(int i = 0 ; i < height[LEVEL].size() ; i++){
		int comp_ind = height[LEVEL][i];
		Gate& compo = component_vector[comp_ind];
		int output_num = compo.size - compo.l_in;
		for(int op = 0 ; op < output_num; op++){
			int signal_id = compo.outputs[op].signal_index;
			Signal& signall = signal_vector[signal_id];

			int part_number = gate_partition_vec[cnt_gate];

			int total_waveform_size = 0;
			if(signall.name == "Fake"){
				for(int j = 0 ; j < part_number ; j++){
					h_partition_output_address[cnt_partition + j] = 0;
				}
				cnt_partition += part_number;
			}
			else{

				for(int j = 0 ; j < part_number ; j++){
					total_waveform_size += h_stimuli_size[cnt_partition + j];
					//cout<<total_waveform_size<<endl;
				}


				//cout<<"!! "<<total_waveform_size<<endl;
				if(total_waveform_size <= 0){
					cout<<signall.name<<" size is 0 "<<compo.name<<" "<<total_waveform_size<<"   /"<<cnt_partition<<endl;
					for(int j = 0 ; j < compo.inputs.size(); j++){
						int signal_id = compo.inputs[j].signal_index;
						Signal& signallf = signal_vector[signal_id];
						int ptr1 = signallf.alloc_ptr_first;
						int ptr2 = signallf.alloc_ptr_last;
						cout<<"lets just remind that "<<signallf.name<<" location is "<<ptr1 <<" to "<<ptr2<<endl;
						kernel_print_waveform_g1<<<1,1>>>(ptr1, ptr2, d_M_EVENT_val, d_M_EVENT_time);
						hipDeviceSynchronize();
					}
					for(int j = 0 ; j < part_number ; j++){
						total_waveform_size = h_stimuli_size[cnt_partition + j];
						cout<<total_waveform_size<<endl;
					}
					exit(0);
				}
				int ptr1 = tree.t_allocate(total_waveform_size);
				if(ptr1 > gpu_size - gpu_size / 8 || ptr1 + total_waveform_size > gpu_size){
					cout<<"sdfsdfsdfsdfsdf "<<signall.name<<" "<<ptr1<<" "<<ptr1 + total_waveform_size<<endl;
				}
				signall.alloc_ptr_first = ptr1;
				signall.alloc_ptr_last = ptr1 + total_waveform_size;

				total_waveform_size = 0;
				for(int j = 0 ; j < part_number ; j++){
					h_partition_output_address[cnt_partition + j] = total_waveform_size + ptr1;
					total_waveform_size += h_stimuli_size[cnt_partition + j];
				}
				cnt_partition += part_number;
			}
			cnt_gate++;
		}
	}
}

__global__ void kernel4_make_saif_g1(int part_n, long long int saif_min, long long int saif_mx, int *d_signal_pointer_first, int *d_signal_pointer_last, bool *d_signal_ending_part, bool *d_fake_part, unsigned int *d_saif_part, int *d_M_EVENT_val, unsigned int *d_M_EVENT_time){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	bool fake_bool = d_fake_part[index];

	if(index < part_n && fake_bool == false){
		int pointer1 = d_signal_pointer_first[index];
		int pointer2 = d_signal_pointer_last[index];
		bool ending = d_signal_ending_part[index];
		long long int mx;
		if(ending){
			mx = saif_mx;
		}
		else{
			mx = d_M_EVENT_time[pointer2];
		}
		//printf("ok we are in part : %d  p1 : %d p2 : %d lowest time : %lld  highest time : %lld\n",index, pointer1, pointer2, d_M_EVENT_time[pointer1], mx);
		long long int x_val = 0;
		long long int i_val = 0;
		long long int o_val = 0;
		for(int i = pointer1 ; i < pointer2; i++){
			long long int t1 = max(saif_min, min(saif_mx,(long long int) d_M_EVENT_time[i]));
			long long int t2 = max(saif_min, min(saif_mx, mx));
			if(i + 1 < pointer2){
				t2 = max(saif_min, min(saif_mx,(long long int) d_M_EVENT_time[i + 1]));
			}
			long long int duration = t2 - t1;
			int val = d_M_EVENT_val[i];
			if(val == 0){
				//printf("adding %lld to %d\n",duration, val );
				o_val += duration;
			}
			else if(val == 1){
				//printf("adding %lld to %d\n",duration, val );
				i_val += duration;
			}
			else{
				//printf("adding %lld to %d\n",duration, val );
				x_val += duration;
			}
		}
		//printf("in partition : %d  oval : %ld  ival: %ld xval: %ld\n",index, o_val, i_val,x_val );
		d_saif_part[index * 3] = o_val;
		d_saif_part[index * 3 + 1] = i_val;
		d_saif_part[index * 3 + 2] = x_val;
	}
}

void memory_deallocate_g1(){
	free(h_gate_start);
	free(h_gate_end);
	free(h_port_start);
	free(h_port_end);
	free(h_partition_number);
	free(h_gate_ind);
	free(h_gate_partition_number);
	free(h_gate_partition_index);
	free(h_stimuli_sdf_first);
	free(h_stimuli_sdf_end);
	free(h_table_id);
	free(h_table);
	free(h_partition_output_address);
	free(h_stimuli_size);
	free(h_sdf_mask);
	free(h_sdf_s_in);
	free(h_t_edge);
	free(h_signal_pointer_first);
	free(h_signal_pointer_last);
	free(h_signal_ending_part);
	free(h_saif_part);
	free(h_fake_signal);
	free(h_fake_part);

	hipFree(d_gate_start);
	hipFree(d_gate_end);
	hipFree(d_port_start);
	hipFree(d_port_end);
	hipFree(d_gate_ind);
	hipFree(d_gate_partition_number);
	hipFree(d_out_pointer);
	hipFree(d_gate_partition_index);
	hipFree(d_port_partition_start);
	hipFree(d_port_partition_end);
	hipFree(d_partition_pre);
	hipFree(d_partition_address_first);
	hipFree(d_partition_address_last);
	hipFree(d_zero_time);
	hipFree(d_partition_number);
	hipFree(d_stimuli_sdf_first);
	hipFree(d_stimuli_sdf_end);
	hipFree(d_table_id);
	hipFree(d_table);
	hipFree(d_tmp_memory_time);
	hipFree(d_partition_output_address);
	hipFree(d_partition_min_time);
	hipFree(d_stimuli_size);
	hipFree(d_sdf_mask);
	hipFree(d_sdf_s_in);
	hipFree(d_t_edge);
	hipFree(d_signal_pointer_last);
	hipFree(d_signal_pointer_first);
	hipFree(d_signal_ending_part);
	hipFree(d_saif_part);
	hipFree(d_fake_signal);
	hipFree(d_fake_part);
}




void saif_initialize_g1(int LEVEL){

	int cnt_partition = 0;
	for(int i = 0 ; i < height[LEVEL].size(); i++){
		int comp_ind = height[LEVEL][i];
		Gate& compo = component_vector[comp_ind];
		int output_num = compo.size - compo.l_in;
		for(int op = 0 ; op < output_num; op++){
			int signal_id = compo.outputs[op].signal_index;
			Signal& signall = signal_vector[signal_id];
			int ptr1 = signall.alloc_ptr_first;
			int ptr2 = signall.alloc_ptr_last;
			//cout<<"lets just remind that "<<signall.name<<" location is "<<ptr1 <<" to "<<ptr2<<endl;
			//kernel_print_waveform_g1<<<1,1>>>(ptr1, ptr2, d_M_EVENT_val, d_M_EVENT_time);
			//hipDeviceSynchronize();
			int sz = ptr2 - ptr1;
			int chunk_sz = sz / sig_part_number;
			for(int j = 0 ; j < sig_part_number; j++){
				if(signall.name == "Fake"){
					h_fake_part[cnt_partition] = true;
				}
				else{
					h_fake_part[cnt_partition] = false;
				}
				h_signal_pointer_first[cnt_partition] = ptr1 + j * chunk_sz;
				h_signal_pointer_last[cnt_partition] = ptr1 + (j + 1) * chunk_sz;
				h_signal_ending_part[cnt_partition] = false;
				if(j == sig_part_number - 1){
					h_signal_pointer_last[cnt_partition] = ptr2;
					h_signal_ending_part[cnt_partition] = true;
				}
				cnt_partition++;
			}
		}
	}
}


void saif_process_g1(int LEVEL){

	int cnt_partition = 0;
	for(int i = 0 ; i < height[LEVEL].size(); i++){
		int comp_ind = height[LEVEL][i];
		Gate& compo = component_vector[comp_ind];
		int output_num = compo.size - compo.l_in;
		for(int op = 0 ; op < output_num; op++){
			int signal_id = compo.outputs[op].signal_index;
			Signal& signall = signal_vector[signal_id];
			if(signall.name == "Fake"){
				cnt_partition += sig_part_number;
				continue;
			}
			//cout<<signall.name<<endl;
			for(int j = 0 ; j < sig_part_number; j++){
				int idx = cnt_partition * 3;
				long long int ov, iv, xv;
				ov = h_saif_part[idx];
				iv = h_saif_part[idx + 1];
				xv = h_saif_part[idx + 2];
				signall.o_val += ov;
				signall.i_val += iv;
				signall.x_val += xv;
				//cout<<"in partition "<<j<<" we have "<<ov<<" "<<iv<<" "<<xv<<endl;
				cnt_partition++;
			}
			//if(i < 5)
			//cout<<" saif : "<<signall.name<<" "<<signall.o_val<<" "<<signall.i_val<<" "<<signall.x_val<<endl;
		}
	}
}

void gpusimultaion1(){

	max_part_number = 40000;
	sig_part_number = 64;
	thread_number = 128;
	gpu_size = 1000*1000*1000;
	cout<<"boom"<<endl;
	input_init_g1();
	cout<<"bam"<<endl;
	kernel0_input_init_g1_g1<<<((total_input_sig * sig_part_number + thread_number - 1) / thread_number), thread_number>>>(sig_part_number, total_input_sig, d_sig_input_M_pointer, d_sig_input_D_pointer, d_sig_input_size, d_input_time, d_input_val, d_M_EVENT_time, d_M_EVENT_val);
	input_free_g1();

	for(int i = 0; i < height.size(); i++){
		//if(i == 1) break;
		cout<<"\n\n--------------------------------------\n\n\n\nlevel "<<i<<endl;
		initialize_level_g1(i);
		memory_allocate_g1();


		set_arrays_g1(i);
		//cout<<"tota num gates : "<<total_number_of_gates<<endl;
		hipMemcpy(d_gate_start, h_gate_start, total_number_of_gates*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_gate_end, h_gate_end, total_number_of_gates*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_port_start, h_port_start, total_number_of_ports*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_port_end, h_port_end, total_number_of_ports*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_gate_partition_index, h_gate_partition_index, total_number_of_gates*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_partition_number, h_partition_number, total_number_of_partitions*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_gate_ind, h_gate_ind, total_number_of_partitions*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_gate_partition_number, h_gate_partition_number, total_number_of_partitions*sizeof(int), hipMemcpyHostToDevice);

		//cout<<"kernel1 "<<total_number_of_partitions<<endl;
		kernel1_find_parttion_bounds_g1<<<((total_number_of_partitions + thread_number - 1) / thread_number), thread_number>>>(total_number_of_partitions, d_gate_partition_number, d_partition_number, d_gate_ind, d_zero_time, d_partition_pre, d_partition_address_first, d_partition_address_last, d_port_partition_start, d_port_partition_end, d_gate_partition_index, d_gate_start,  d_gate_end, d_port_start, d_port_end, d_M_EVENT_time, d_M_EVENT_val);
		//hipDeviceSynchronize();

		hipMemcpy(d_stimuli_sdf_first, h_stimuli_sdf_first, total_number_of_partitions*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_stimuli_sdf_end, h_stimuli_sdf_end, total_number_of_partitions*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_table_id, h_table_id, total_number_of_partitions*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_table, h_table, (t_ans.size())*sizeof(int) , hipMemcpyHostToDevice);
		hipMemcpy(d_sdf_mask, h_sdf_mask, total_sum_of_sdf*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_sdf_s_in, h_sdf_s_in, (total_sum_of_sdf / 8)*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_t_edge, h_t_edge, 16*sizeof(int8_t), hipMemcpyHostToDevice);
		//cout<<"A "<<total_number_of_partitions<<endl;

		//cout<<"kernel2"<<endl;
		kernel2_find_partition_sizes_g1<<<((total_number_of_partitions + thread_number - 1) / thread_number), thread_number>>>(thread_number, total_number_of_partitions, d_out_pointer, d_gate_partition_number, MX, d_partition_number, d_zero_time, d_partition_min_time, d_stimuli_size, d_partition_pre, d_stimuli_sdf_first, d_stimuli_sdf_end, d_partition_address_first, d_partition_address_last, d_port_partition_start, d_port_partition_end, d_sdf_mask, d_sdf_s_in, d_t_edge, d_table_id, d_tmp_memory_time, d_table, d_M_EVENT_val, d_M_EVENT_time);
		//hipDeviceSynchronize();
		kernel2_2_find_partition_sizes_g1<<<((total_number_of_partitions + thread_number - 1) / thread_number), thread_number>>>( total_number_of_partitions, d_out_pointer, d_gate_partition_number, MX, d_partition_number, d_zero_time, d_partition_min_time,  d_stimuli_size, d_partition_pre, d_stimuli_sdf_first, d_stimuli_sdf_end, d_partition_address_first, d_partition_address_last, d_port_partition_start, d_port_partition_end, d_t_edge, d_table_id, d_tmp_memory_time, d_table, d_M_EVENT_val, d_M_EVENT_time);
		//cout<<"end kernel2"<<endl;
		hipMemcpy(h_stimuli_size, d_stimuli_size, total_number_of_partitions*sizeof(int), hipMemcpyDeviceToHost);

		cout<<"tree allocation of level "<<i<<endl;
		tree_allocations_g1(i);

		hipMemcpy(d_partition_output_address, h_partition_output_address, total_number_of_partitions*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_fake_signal, h_fake_signal, total_number_of_partitions*sizeof(bool), hipMemcpyHostToDevice);

		cout<<"kernel3"<<endl;
		kernel3_gpu_single_gate_simulation_g1<<<((total_number_of_partitions + thread_number - 1) / thread_number), thread_number>>>(thread_number, total_number_of_partitions, MX, d_partition_number, d_partition_output_address, d_zero_time, d_partition_min_time, d_stimuli_size, d_partition_pre, d_stimuli_sdf_first, d_stimuli_sdf_end, d_partition_address_first, d_partition_address_last, d_port_partition_start, d_port_partition_end, d_sdf_mask, d_sdf_s_in, d_t_edge, d_table_id, d_table, d_M_EVENT_val, d_M_EVENT_time, d_fake_signal);
		hipDeviceSynchronize();

		saif_initialize_g1(i);

		hipMemcpy(d_signal_pointer_first, h_signal_pointer_first, total_number_of_signal_parts*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_signal_pointer_last, h_signal_pointer_last, total_number_of_signal_parts*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_signal_ending_part, h_signal_ending_part, total_number_of_signal_parts*sizeof(bool),hipMemcpyHostToDevice);
		hipMemcpy(d_fake_part, h_fake_part, total_number_of_signal_parts*sizeof(bool), hipMemcpyHostToDevice);

		cout<<"kernel4"<<endl;
		kernel4_make_saif_g1<<<((total_number_of_signal_parts + thread_number - 1) / thread_number), thread_number>>>(total_number_of_signal_parts, MN, MX, d_signal_pointer_first, d_signal_pointer_last, d_signal_ending_part, d_fake_part, d_saif_part, d_M_EVENT_val, d_M_EVENT_time);
		hipDeviceSynchronize();

		hipMemcpy(h_saif_part, d_saif_part, total_number_of_signal_parts*3*sizeof(unsigned int),hipMemcpyDeviceToHost);
		saif_process_g1(i);

		//if(vcd_make){
		//	make_vcd_output();
		//}

		memory_deallocate_g1();
	}

	hipFree(d_M_EVENT_val);
	hipFree(d_M_EVENT_time);
}
